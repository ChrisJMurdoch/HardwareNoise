#include "hip/hip_runtime.h"

// This has to be directly included into a translation unit as it contains
// device code, wrap include statement in a namespace to avoid linker errors.

#pragma once

__host__ __device__
int intHash(int x)
{
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    return x;
}

__host__ __device__
float floatHash(int x)
{
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = ((x >> 16) ^ x) * 0x45d9f3b;
    x = (x >> 16) ^ x;
    return ( x % 10000 ) / 9999.0f;
}

__host__ __device__
int combine(int x, int y) {
    return (x*12345) + y;
}

__host__ __device__
float lerp(float a, float b, float x)
{
    return a + x * (b - a);
}

__host__ __device__
float diverge(float x)
{
    const float PI = 3.14159265358979323846;
    return 0.5 - ( cos( fmod(x,1.0f) * PI )*0.5 );
}

__host__ __device__
float squash(float x)
{
    const float PI = 3.14159265358979323846;
    return acos( -2*(x-0.5) ) / PI;
}

__host__ __device__
float falloff(float x)
{
    const float PI = 3.14159265358979323846;
    return powf( sin(x*PI), 0.05 );
}

__host__ __device__
float step(float x, float a, float s)
{
    return ( ( floor(x*s) + powf( diverge(x*s), a ) ) / s ) + ( 1 / (2*s) );
}

__host__ __device__
float elevate(float x)
{
    const float A = 0.6;
    return ( 1 - A ) + A * powf( x, 2 );
}

// SAMPLES (X,Y,P) => Z

__host__ __device__
float hashSample(int x, int y, float period)
{
    return floatHash( combine(x, y) );
}

__host__ __device__
float sinSample(int x, int y, float period)
{
    const float PI = 3.14159265358979323846;
    float xd = ( sin( x * (2*PI) / period ) + 1 ) / 2;
    float yd = ( sin( y * (2*PI) / period ) + 1 ) / 2;
    return xd * yd;
}

__host__ __device__
float perlinSample(int x, int y, float period)
{
    // Square coords
    int X = std::floor( x / period );
    int Y = std::floor( y / period );

    // Relative point coords
    float rx = (x/period) - X;
    float ry = (y/period) - Y;

    // Square corner vectors
    glm::vec2 BL = glm::normalize( glm::vec2( floatHash( combine( X , Y ) )-0.5, floatHash( combine( X , Y )+1 )-0.5 ) );
    glm::vec2 BR = glm::normalize( glm::vec2( floatHash( combine(X+1, Y ) )-0.5, floatHash( combine(X+1, Y )+1 )-0.5 ) );
    glm::vec2 TL = glm::normalize( glm::vec2( floatHash( combine( X ,Y+1) )-0.5, floatHash( combine( X ,Y+1)+1 )-0.5 ) );
    glm::vec2 TR = glm::normalize( glm::vec2( floatHash( combine(X+1,Y+1) )-0.5, floatHash( combine(X+1,Y+1)+1 )-0.5 ) );

    // Relational vectors
    glm::vec2 point = glm::vec2( rx, ry );
    glm::vec2 BLr = glm::vec2( 0, 0 ) - point;
    glm::vec2 BRr = glm::vec2( 1, 0 ) - point;
    glm::vec2 TLr = glm::vec2( 0, 1 ) - point;
    glm::vec2 TRr = glm::vec2( 1, 1 ) - point;

    // Dot products
    float BLd = glm::dot( BL, BLr );
    float BRd = glm::dot( BR, BRr );
    float TLd = glm::dot( TL, TLr );
    float TRd = glm::dot( TR, TRr );

    // Interpolate using diverge
    float bottom = lerp( BLd, BRd, diverge(point.x) );
    float top = lerp( TLd, TRd, diverge(point.x) );
    float centre = lerp( bottom, top, diverge(point.y) );

    // 0-1
    return (centre+1) / 2;
}

__host__ __device__
float perlinRidgeSample(int x, int y, float period)
{
    float neg = ( perlinSample(x, y, period)*2 ) - 1 ;
    return 0.6 - abs( neg );
}

__host__ __device__
float perlinCutSample(int x, int y, float period)
{
    float neg = ( perlinSample(x, y, period)*2 ) - 1 ;
    return abs( neg );
}

// SAMPLE COMPOSITES

__host__ __device__
float fractal(int x, int y, float period, MathEngine::Sample sample, int octaves)
{
    // Octaves
    float height = 0;
    float max = 0;
    for (int o=0; o<octaves; o++)
    {
        // Caluculate amplitude and period
        const float lacunarity = 0.5, persistance = 0.4;
        float pmult = pow(lacunarity, o), amplitude = pow(persistance, o);

        // Get sample value
        switch ( sample )
        {
        case MathEngine::hash:
            height += hashSample( x, y, pmult*period ) * amplitude;
            break;
        case MathEngine::sin:
            height += sinSample( x, y, pmult*period ) * amplitude;
            break;
        case MathEngine::perlin:
            height += perlinSample( x, y, pmult*period ) * amplitude;
            break;
        case MathEngine::perlinRidge:
            height += perlinRidgeSample( x, y, pmult*period ) * amplitude;
            break;
        default:
            height += hashSample( x, y, pmult*period ) * amplitude;
            break;
        }
        max += amplitude;
    }
    return height / max;
}

__host__ __device__
float mountain(int x, int y, float period)
{
    // Domain distortion
    float distortion = period;
    float dx = x + perlinSample(x, y, period/1) * distortion;
    float dy = y + perlinSample(x+9999, y+9999, period/2) * distortion;

    // Amplitudes
    float a1 = 32;
    float a2 = 16;
    float a3 = 8;
    float a4 = 4;
    float a5 = 2;
    float a6 = 1;

    // Terrain samples
    float s1 = perlinSample(dx, dy, period/ 1);
    float s2 = perlinSample(dx, dy, period/ 2);
    float s3 = perlinSample(dx, dy, period/ 4);
    float s4 = perlinSample(x, y, period/ 8);
    float s5 = perlinSample(x, y, period/16);
    float s6 = perlinSample(x, y, period/32);

    // Merge
    float amp = a1 + a2 + a3 + a4 + a5 + a6;
    float total = ( (s1*a1) + (s2*a2) + (s3*a3) + (s4*a4) + (s5*a5) + (s6*a6) ) / amp;
    return total;
}

__host__ __device__
float plateau(int x, int y, float period)
{
    // Domain distortion
    float distortion = 50;
    float dx = x + perlinSample(x, y, period/1) * distortion;
    float dy = y + perlinSample(x+9999, y+9999, period/2) * distortion;

    // Amplitudes
    float a1 = 32;
    float a2 = 16;
    float a3 = 8;
    float a4 = 4;
    float a5 = 0;
    float a6 = 0;

    // Terrain samples
    float s1 = perlinSample(dx, dy, period/ 1);
    float s2 = perlinSample(dx, dy, period/ 2);
    float s3 = perlinSample(dx, dy, period/ 4);
    float s4 = perlinSample(dx, dy, period/ 8);
    float s5 = perlinSample(x, y, period/16);
    float s6 = perlinSample(x, y, period/32);

    // Merge
    float amp = a1 + a2 + a3 + a4 + a5 + a6;
    float stepped = ( (s1*a1) + (s2*a2) + (s3*a3) + (s4*a4) + (s5*a5) + (s6*a6) ) / amp;
    float unstepped = ( 0 ) / amp;
    return ( step( diverge(stepped), 10, 20) + unstepped );
}
