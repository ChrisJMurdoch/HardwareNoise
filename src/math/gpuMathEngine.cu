#include "hip/hip_runtime.h"

#include <math/gpuMathEngine.hpp>

#include <utility/log.hpp>

#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
namespace gpucommon
{
    #include "common.cu"
}

// HARDWARE SETTINGS
#define STREAMS 12
#define WARPS 16

// DEVICE SETUP

GPUMathEngine::GPUMathEngine()
{
    cudaCheck( hipDeviceGetAttribute(&nSM, hipDeviceAttributeMultiprocessorCount, 0) );
}

// HEIGHTMAP GENERATION

__global__
void heightmapKernel(float *out, int dimension, float min, float max, GPUMathEngine::Sample sample, float period, int octaves )
{
    // Start index
    int startIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Grid stride
    int index = startIndex;

    // Thread calculation
    int x = index % dimension;
    int y = index / dimension;
    do
    {
        if ( x==0 || x==dimension-1 || y==0 || y==dimension-1 )
        {
            out[index] = -10;
            index += blockDim.x*gridDim.x;
            x = index % dimension;
            y = index / dimension;
            continue;
        }

        // Get sample
        float value;
        switch ( sample )
        {
        case GPUMathEngine::mountain:
            value = gpucommon::mountain(x, y, period);
            break;
        default:
            value = gpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        out[index] = min + ( value * (max-min) );

        // Stride forward
        index += blockDim.x*gridDim.x;
        x = index % dimension;
        y = index / dimension;
    }
    while ( y<dimension );
}

void GPUMathEngine::generateHeightMap(float *out, int dimension, float min, float max, Sample sample, float period, int octaves)
{
    // Allocate device memory
    float *d_out;
    int size = dimension*dimension*sizeof(float);
    cudaCheck( hipMalloc( (void **)&d_out, size ) );
    heightmapKernel<<<nSM, WARPS*32>>>( d_out, dimension, min, max, sample, period, octaves );
    cudaCheck( hipMemcpy(out, d_out, size, hipMemcpyDeviceToHost) );
    cudaCheck( hipFree(d_out) );
}

void GPUMathEngine::brush(float *map, int width, int x, int y, float amount, int radius)
{
    int dim = (radius*2)+1;
    amount /= (float)dim*dim;

    // Sum up eligible cell distribution
    float total = 0;
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;
        
        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        total += close;
    }

    // Calculate total distribution
    float mult = dim*dim / total;

    // Add values
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;

        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        // Alter cell
        map[ x+xo + ((y+yo)*width) ] += mult*amount*close;
        total++;
    }
}

float GPUMathEngine::getCellHeight(float *map, int width, int x, int y)
{
    // Simulate walls on edge of map to avoid out of bounds erosion
    return ( x<0 || x>=width || y<0 || y>=width ) ? 9999 : map[ x + (y*width) ]+1000;
}

void GPUMathEngine::erodeCell(float *map, int width, int x, int y, float speed, float sediment, int radius)
{
    // Get cell height
    float height = getCellHeight(map, width, x, y);

    // Find lowest cell in 3x3 (including self)
    int lx=x, ly=y;
    float lh = height;
    for (int xo=-1; xo<2; xo++) for (int yo=-1; yo<2; yo++)
    {
        float h = getCellHeight(map, width, x+xo, y+yo);
        if ( h<lh )
        {
            lx = x+xo;
            ly = y+yo;
            lh = h;
        }
    }

    // Calculate difference
    float delta = height - lh;

    // Base case => Not enough speed to move
    if ( -delta >= speed )
    {
        brush(map, width, x, y, sediment, radius);
        return;
    }

    // Calculate new speed with friction
    const float FRICTION = 0.1f;
    speed += delta - FRICTION;
    
    // Calculate new sediment capacity
    float capacity = speed;
    float deposit = sediment - capacity;

    // Add / Remove sediment from surrounding cells
    brush(map, width, x, y, deposit, radius);

    // Recurse on downhill cell
    erodeCell(map, width, lx, ly, speed, capacity, radius);
}

void GPUMathEngine::erode(float *map, int width, int droplets, int radius)
{
    static float s_i = 0;
    srand( s_i++ );
    for (int i=0; i<droplets; i++)
    {
        int x = rand() % width, y = rand() % width;
        erodeCell(map, width, x, y, 0, 0, radius);
    }
}

// MACROS

inline void GPUMathEngine::cudaCheck(hipError_t err)
{
    if (err != hipSuccess)
    {
        Log::print( Log::error, "CudaCheck:" );
        Log::print( Log::error, hipGetErrorString(err) );
    }
}
