#include "hip/hip_runtime.h"

#include <math/gpuMathEngine.hpp>

#include <utility/log.hpp>

#include "hip/hip_runtime.h"
#define GLM_FORCE_CUDA
#include <glm/glm.hpp>
namespace gpucommon
{
    #include "common.cu"
}

// Hardware settings
#define STREAMS 12
#define WARPS 16

// DEVICE SETUP

GPUMathEngine::GPUMathEngine()
{
    cudaCheck( hipDeviceGetAttribute(&nSM, hipDeviceAttributeMultiprocessorCount, 0) );
}

// HEIGHTMAP GENERATION

__global__
void heightmapKernel(float *out, int dimension, float min, float max, GPUMathEngine::Sample sample, float period, int octaves )
{
    // Start index
    int startIndex = threadIdx.x + blockIdx.x * blockDim.x;

    // Grid stride
    int index = startIndex;

    // Thread calculation
    int x = index % dimension;
    int y = index / dimension;
    do
    {
        if ( x==0 || x==dimension-1 || y==0 || y==dimension-1 )
        {
            out[index] = -10;
            index += blockDim.x*gridDim.x;
            x = index % dimension;
            y = index / dimension;
            continue;
        }

        // Get sample
        float value;
        switch ( sample )
        {
        case GPUMathEngine::mountain:
            value = gpucommon::mountain(x, y, period);
            break;
        default:
            value = gpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        out[index] = min + ( value * (max-min) );

        // Stride forward
        index += blockDim.x*gridDim.x;
        x = index % dimension;
        y = index / dimension;
    }
    while ( y<dimension );
}

Heightmap GPUMathEngine::generateHeightMap(std::map<std::string, std::string> &settings, int dimension, float xOff, float yOff )
{
    float min = stof(settings["min"]),
          max = stof(settings["max"]);
    int period = stoi(settings["period"]),
        octaves = stoi(settings["octaves"]);
    Sample sample = getSample( settings["sampling"] );
    float *nodes = new float[dimension*dimension];

    // Allocate device memory
    float *d_out;
    int size = dimension*dimension*sizeof(float);
    cudaCheck( hipMalloc( (void **)&d_out, size ) );
    heightmapKernel<<<nSM, WARPS*32>>>( d_out, dimension, min, max, sample, period, octaves );
    cudaCheck( hipMemcpy(nodes, d_out, size, hipMemcpyDeviceToHost) );
    cudaCheck( hipFree(d_out) );

    return Heightmap(&nodes, dimension);
}

// MACROS

inline void GPUMathEngine::cudaCheck(hipError_t err)
{
    if (err != hipSuccess)
    {
        Log::println( Log::error, "CudaCheck:" );
        Log::println( Log::error, hipGetErrorString(err) );
    }
}
