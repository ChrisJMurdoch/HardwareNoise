#include "hip/hip_runtime.h"

#include <math/cpuMathEngine.hpp>

#include <glm/glm.hpp>
namespace cpucommon
{
    #include "common.cu"
}

#include <utility/log.hpp>

// FUNCTIONS

Heightmap CPUMathEngine::generateHeightMap(std::map<std::string, std::string> &settings, int dimension, float xOff, float yOff )
{
    // Get settings
    float min = stof(settings["min"]),
          max = stof(settings["max"]);
    int period = stoi(settings["period"]),
        octaves = stoi(settings["octaves"]);
    Sample sample = getSample( settings["sampling"] );
    float *nodes = new float[dimension*dimension];
    
    for (int y=0; y<dimension; y++) for (int x=0; x<dimension; x++)
    {
        if ( x==0 || x==dimension-1 || y==0 || y==dimension-1 )
        {
            nodes[y*dimension + x] = -10;
            continue;
        }

        // Custom sampling
        float value;
        switch ( sample )
        {
        case mountain:
            value = cpucommon::mountain(x, y, period);
            break;
        default:
            value = cpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        
        nodes[y*dimension + x] = min + ( value * (max-min) );
    }

    return Heightmap(&nodes, dimension);
}
