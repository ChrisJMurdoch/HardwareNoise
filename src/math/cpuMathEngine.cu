#include "hip/hip_runtime.h"

#include <math/cpuMathEngine.hpp>

#include <glm/glm.hpp>
namespace cpucommon
{
    #include "common.cu"
}

// FUNCTIONS

void CPUMathEngine::generateHeightMap(float *out, int dimension, float min, float max, Sample sample, float period, int octaves)
{
    for (int y=0; y<dimension; y++) for (int x=0; x<dimension; x++)
    {
        if ( x==0 || x==dimension-1 || y==0 || y==dimension-1 )
        {
            out[y*dimension + x] = -10;
            continue;
        }

        float value;
        // Custom sampling
        switch ( sample )
        {
        case mountain:
            value = cpucommon::mountain(x, y, period);
            break;
        default:
            value = cpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        out[y*dimension + x] = min + ( value * (max-min) );
    }
}

void CPUMathEngine::brush(float *map, int width, int x, int y, float amount, int radius)
{
    int dim = (radius*2)+1;
    amount /= (float)dim*dim;

    // Sum up eligible cell distribution
    float total = 0;
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;
        
        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        total += close;
    }

    // Calculate total distribution
    float mult = dim*dim / total;

    // Add values
    for (int xo=-radius; xo<radius+1; xo++) for (int yo=-radius; yo<radius+1; yo++)
    {
        // Out of bounds
        if  ( x+xo<0 || x+xo>=width || y+yo<0 || y+yo>=width )
            continue;

        // Fade
        float dist = pow( pow( (xo), 2) + pow( (yo), 2), 0.5);
        float close = (radius - dist) / radius * 2;
        close = close<0 ? 0 : close;

        // Alter cell
        map[ x+xo + ((y+yo)*width) ] += mult*amount*close;
        total++;
    }
}

float CPUMathEngine::getCellHeight(float *map, int width, int x, int y)
{
    // Simulate walls on edge of map to avoid out of bounds erosion
    return ( x<0 || x>=width || y<0 || y>=width ) ? 9999 : map[ x + (y*width) ]+1000;
}

void CPUMathEngine::erodeCell(float *map, int width, int x, int y, float speed, float sediment, int radius)
{
    // Get cell height
    float height = getCellHeight(map, width, x, y);

    // Find lowest cell in 3x3 (including self)
    int lx=x, ly=y;
    float lh = height;
    for (int xo=-1; xo<2; xo++) for (int yo=-1; yo<2; yo++)
    {
        float h = getCellHeight(map, width, x+xo, y+yo);
        if ( h<lh )
        {
            lx = x+xo;
            ly = y+yo;
            lh = h;
        }
    }

    // Calculate difference
    float delta = height - lh;

    // Base case => Not enough speed to move
    if ( -delta >= speed )
    {
        brush(map, width, x, y, sediment, radius);
        return;
    }

    // Calculate new speed with friction
    const float FRICTION = 0.1f;
    speed += delta - FRICTION;
    
    // Calculate new sediment capacity
    float capacity = speed;
    float deposit = sediment - capacity;

    // Add / Remove sediment from surrounding cells
    brush(map, width, x, y, deposit, radius);

    // Recurse on downhill cell
    erodeCell(map, width, lx, ly, speed, capacity, radius);
}

void CPUMathEngine::erode(float *map, int width, int droplets, int radius)
{
    static float s_i = 0;
    srand( s_i++ );
    for (int i=0; i<droplets; i++)
    {
        int x = rand() % width, y = rand() % width;
        erodeCell(map, width, x, y, 0, 0, radius);
    }
}
