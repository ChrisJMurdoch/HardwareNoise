#include "hip/hip_runtime.h"

#include <noise/cpuMathEngine.hpp>

#include <glm/glm.hpp>
namespace cpucommon
{
    #include "common.cu"
}

#include <utility/log.hpp>

// FUNCTIONS

Matrix CPUMathEngine::generateHeightMap(std::map<std::string, std::string> &settings, int dimension, float xOff, float yOff )
{
    // Get settings
    float min = stof(settings["min"]),
          max = stof(settings["max"]);
    int period = stoi(settings["period"]),
        octaves = stoi(settings["octaves"]);
    Sample sample = getSample( settings["sampling"] );

    Matrix hm(dimension, dimension);
    
    for (int y=0; y<dimension; y++) for (int x=0; x<dimension; x++)
    {
        if ( x==0 || x==dimension-1 || y==0 || y==dimension-1 )
        {
            hm[y][x] = -10;
            continue;
        }

        // Custom sampling
        float value;
        switch ( sample )
        {
        case mountain:
            value = cpucommon::mountain(x, y, period);
            break;
        case plateau:
            value = cpucommon::plateau(x, y, period);
            break;
        default:
            value = cpucommon::fractal(x, y, period, sample, octaves);
            break;
        }
        
        hm[y][x] = min + ( value * (max-min) );
    }

    return hm;
}
